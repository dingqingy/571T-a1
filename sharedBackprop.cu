#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>


// forward propogation
/*
def forwardBackward(X, y, W, v):
    Z_trans = relu(W@X.T) # mat-mat
    Z = Z_trans.T # trans
    yhat = Z@v # mat-vec
    
    error = y - yhat
    grad_v = Z.T @ error  # mat-vector
    grad_Z = np.outer(error, v) # outer product
    grad_p = dRelu(dZ, Z)
    grad_W = dp.T @ X  # mat-mat
*/

/* Parameter Setup */
#define N 32 // # of input samples
#define D 32 // # of input neurons
#define K 32// # of hidden neurons
#define STEP 0.001 // learning rate or step size

// X: input matrix (n * d)
#define X_HEIGHT N
#define X_WIDTH D
#define X_N X_HEIGHT * X_WIDTH
// Z: ifmap matrix (n * k)
#define Z_HEIGHT N
#define Z_WIDTH K
#define Z_N Z_HEIGHT * Z_WIDTH 
// W: layer 1 weights (k * d)
#define W_HEIGHT K
#define W_WIDTH D
#define W_N W_HEIGHT * W_WIDTH
// v: layer 2 weights
#define V_HEIGHT K
#define V_WIDTH 1
#define V_N V_HEIGHT * V_WIDTH

#define BLOCK_SIZE 32
#define LINEAR_BLOCK_SIZE BLOCK_SIZE * BLOCK_SIZE
#define MAX_ERR 1e-6

__global__ void matrix_mul_shared(double *d_C, double *d_A, double *d_B, int d_a_height, int d_a_width, int d_b_width) {
    // global position in the C (output) matrix
    int cid = blockIdx.y * blockDim.y + threadIdx.y;
    int rid = blockIdx.x * blockDim.x + threadIdx.x;

    // thread position in the block
    int c_thread = threadIdx.y;
    int r_thread = threadIdx.x;

    double sum = 0.0;

    __shared__ double d_A_sub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double d_B_sub[BLOCK_SIZE][BLOCK_SIZE];

    // iterate over tiles across horizontal direction of A
    for(int k = 0; k<(d_a_width-1)/BLOCK_SIZE + 1; k++){
        // load d_A[rid, k*BLOCK_SIZE+c_thread] into d_A_sub[r_thread][c_thread]
        if((rid < d_a_height) && (k*BLOCK_SIZE+c_thread < d_a_width)){
            d_A_sub[r_thread][c_thread] = d_A[rid*d_a_width + k*BLOCK_SIZE+c_thread];
        } else {
            d_A_sub[r_thread][c_thread] = 0.0;
        }

        // load d_B[k*BLOCK_SIZE + r_thread, cid] into d_B_sub[r_thread][c_thread]
        if((k*BLOCK_SIZE+r_thread < d_a_width) && (cid < d_b_width)){
            d_B_sub[r_thread][c_thread] = d_B[(k*BLOCK_SIZE+r_thread)*d_b_width + cid];
        } else {
            d_B_sub[r_thread][c_thread] = 0.0;
        }

        __syncthreads();

        // dot product within a tile
        for(int i = 0; i<d_a_width; i++){
            sum += d_A_sub[r_thread][i] * d_B_sub[i][c_thread];
        }

        __syncthreads();
    }

    if(rid < d_a_height && cid < d_b_width)
        d_C[rid * d_b_width + cid] = sum;
}

__global__ void relu_matrix_mul_shared(double *d_C, double *d_A, double *d_B, int d_a_height, int d_a_width, int d_b_width) {
    // global position in the C (output) matrix
    int cid = blockIdx.y * blockDim.y + threadIdx.y;
    int rid = blockIdx.x * blockDim.x + threadIdx.x;

    // thread position in the block
    int c_thread = threadIdx.y;
    int r_thread = threadIdx.x;

    double sum = 0.0;

    __shared__ double d_A_sub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double d_B_sub[BLOCK_SIZE][BLOCK_SIZE];

    // iterate over tiles across horizontal direction of A
    for(int k = 0; k<(d_a_width-1)/BLOCK_SIZE + 1; k++){
        // load d_A[rid, k*BLOCK_SIZE+c_thread] into d_A_sub[r_thread][c_thread]
        if((rid < d_a_height) && (k*BLOCK_SIZE+c_thread < d_a_width)){
            d_A_sub[r_thread][c_thread] = d_A[rid*d_a_width + k*BLOCK_SIZE+c_thread];
        } else {
            d_A_sub[r_thread][c_thread] = 0.0;
        }

        // load d_B[k*BLOCK_SIZE + r_thread, cid] into d_B_sub[r_thread][c_thread]
        if((k*BLOCK_SIZE+r_thread < d_a_width) && (cid < d_b_width)){
            d_B_sub[r_thread][c_thread] = d_B[(k*BLOCK_SIZE+r_thread)*d_b_width + cid];
        } else {
            d_B_sub[r_thread][c_thread] = 0.0;
        }

        __syncthreads();

        // dot product within a tile
        for(int i = 0; i<d_a_width; i++){
            sum += d_A_sub[r_thread][i] * d_B_sub[i][c_thread];
        }

        __syncthreads();
    }

    if(rid < d_a_height && cid < d_b_width)
        d_C[rid * d_b_width + cid] = (sum>0)?sum:0;
}

__global__ void d_relu_matrix_mul_shared(double *d_C, double *d_A, double *d_B, double *d_act, int d_a_height, int d_a_width, int d_b_width) {
    // global position in the C (output) matrix
    int cid = blockIdx.y * blockDim.y + threadIdx.y;
    int rid = blockIdx.x * blockDim.x + threadIdx.x;

    // thread position in the block
    int c_thread = threadIdx.y;
    int r_thread = threadIdx.x;

    double sum = 0.0;

    __shared__ double d_A_sub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double d_B_sub[BLOCK_SIZE][BLOCK_SIZE];

    // iterate over tiles across horizontal direction of A
    for(int k = 0; k<(d_a_width-1)/BLOCK_SIZE + 1; k++){
        // load d_A[rid, k*BLOCK_SIZE+c_thread] into d_A_sub[r_thread][c_thread]
        if((rid < d_a_height) && (k*BLOCK_SIZE+c_thread < d_a_width)){
            d_A_sub[r_thread][c_thread] = d_A[rid*d_a_width + k*BLOCK_SIZE+c_thread];
        } else {
            d_A_sub[r_thread][c_thread] = 0.0;
        }

        // load d_B[k*BLOCK_SIZE + r_thread, cid] into d_B_sub[r_thread][c_thread]
        if((k*BLOCK_SIZE+r_thread < d_a_width) && (cid < d_b_width)){
            d_B_sub[r_thread][c_thread] = d_B[(k*BLOCK_SIZE+r_thread)*d_b_width + cid];
        } else {
            d_B_sub[r_thread][c_thread] = 0.0;
        }

        __syncthreads();

        // dot product within a tile
        for(int i = 0; i<d_a_width; i++){
            sum += d_A_sub[r_thread][i] * d_B_sub[i][c_thread];
        }

        __syncthreads();
    }

    if(rid < d_a_height && cid < d_b_width)
        d_C[rid * d_b_width + cid] = (d_act[rid * d_b_width + cid]>0)?sum:0;
}


__global__ void matrix_transpose(double *d_out, double *d_in, int d_in_width, int d_out_width) {
    int cid = blockIdx.y * blockDim.y + threadIdx.y;
    int rid = blockIdx.x * blockDim.x + threadIdx.x;

    if(cid < d_in_width && rid < d_out_width){
        d_out[cid * d_out_width + rid] = d_in[rid * d_in_width + cid];
    }
}

__global__ void vector_sub(double *out, double *a, double *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n){
        out[tid] = a[tid] - b[tid];
    }
}

__global__ void update(double *d_weights, double *d_grads, double step, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n){
        d_weights[tid] -= step * d_grads[tid];
    }
}

__global__ void square(double *out, double *in,  int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n){
        out[tid] = in[tid] * in[tid];
    }
}
int main(){
    // forward variables
    double *h_X, *h_W, *h_Z, *h_Z_T, *h_v, *h_yhat, *h_y;
    double *d_X, *d_X_T, *d_W, *d_Z, *d_Z_T, *d_v, *d_yhat, *d_y;

    // backward variables
    double *h_error, *h_grad_v, *h_grad_Z, *h_grad_p_T, *h_grad_W, *h_err_sq;
    double *d_error, *d_grad_v, *d_grad_Z, *d_grad_p_T, *d_grad_W, *d_err_sq;
    // double *h_ref; // compute verified results
    // Allocate host memory
    h_X = (double*)malloc(sizeof(double) * X_N);
    h_W = (double*)malloc(sizeof(double) * W_N);
    h_v = (double*)malloc(sizeof(double) * V_N);
    h_Z_T = (double*)malloc(sizeof(double) * Z_N);
    h_Z = (double*)malloc(sizeof(double) * Z_N);
    h_yhat = (double*)malloc(sizeof(double) * N);
    h_y = (double*)malloc(sizeof(double) * N);
    h_error = (double*)malloc(sizeof(double) * N);
    h_grad_v = (double*)malloc(sizeof(double) * V_N);
    h_grad_Z = (double*)malloc(sizeof(double) * Z_N);
    h_grad_p_T = (double*)malloc(sizeof(double) * Z_N);
    h_grad_W = (double*)malloc(sizeof(double) * W_N);
    h_err_sq = (double*)malloc(sizeof(double) * N);
    // h_ref = (double*)malloc(sizeof(double) * N);

    // Initialize host arrays
    
    /***       TEST 1    ***/
    /*
    for(int i = 0; i < X_N; i++){
        if(i == 1 || i == 3){
            h_X[i] = (double)(-i-1);
        } else{
            h_X[i] = (double)(i+1);
        }
    }
    for(int i = 0; i < W_N; i++){
        h_W[i] = double(i+1);
    }
    for(int i = 0; i < V_HEIGHT; i++){
        h_v[i] = (double)(i+1);
    }
    for(int i = 0; i < N; i++){
        h_y[i] = (double)(i+1);
    }
    */
    
    /***       TEST 2    ***/
    srand((unsigned int)time(NULL));
    // random uniform from [-a, a]
    double a = 1.0;
    for (int i = 0; i< X_N; i++){
        h_X[i] = -a + (double)rand()/(double)(RAND_MAX)*a;
    }
    for (int i = 0; i< W_N; i++){
        h_W[i] = -a + (double)rand()/(double)(RAND_MAX)*a;
    }

    for (int i = 0; i< V_N; i++){
        h_v[i] = -a + (double)rand()/(double)(RAND_MAX)*a;
    }
    for (int i = 0; i< N; i++){
        h_y[i] = -a + (double)rand()/(double)(RAND_MAX)*a;
    }

    // Allocate device memory
    hipMalloc((void**)&d_X, sizeof(double) * X_N);
    hipMalloc((void**)&d_X_T, sizeof(double) * X_N);
    hipMalloc((void**)&d_Z, sizeof(double) * Z_N);
    hipMalloc((void**)&d_Z_T, sizeof(double) * Z_N);
    hipMalloc((void**)&d_W, sizeof(double) * W_N);
    hipMalloc((void**)&d_v, sizeof(double) * V_N);
    hipMalloc((void**)&d_yhat, sizeof(double) * N);
    hipMalloc((void**)&d_y, sizeof(double) * N);
    hipMalloc((void**)&d_error, sizeof(double) * N);
    hipMalloc((void**)&d_grad_v, sizeof(double) * V_N);
    hipMalloc((void**)&d_grad_Z, sizeof(double) * Z_N);
    hipMalloc((void**)&d_grad_p_T, sizeof(double) * Z_N);
    hipMalloc((void**)&d_grad_W, sizeof(double) * W_N);
    hipMalloc((void**)&d_err_sq, sizeof(double) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_X, h_X, sizeof(double) * X_N, hipMemcpyHostToDevice);
    hipMemcpy(d_W, h_W, sizeof(double) * W_N, hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, sizeof(double) * V_N, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, sizeof(double) * N, hipMemcpyHostToDevice);

    int iters = 20;
    for (int i = 0; i < iters; i++){
        // Executing kernel
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        // X_HEIGHT (N) corresponding to OUT_WIDTH, X_WIDTH (D) corresponding to IN_WIDTH
        dim3 dimGrid1(N / BLOCK_SIZE + 1,D / BLOCK_SIZE + 1);
        matrix_transpose<<<dimGrid1,dimBlock>>>(d_X_T, d_X, D, N);
        
        dim3 dimGrid2(K / BLOCK_SIZE + 1, N / BLOCK_SIZE + 1);
        relu_matrix_mul_shared<<<dimGrid2,dimBlock>>>(d_Z_T, d_W, d_X_T, K, D, N);
        
        dim3 dimGrid3(K / BLOCK_SIZE + 1, N / BLOCK_SIZE + 1);
        matrix_transpose<<<dimGrid3,dimBlock>>>(d_Z, d_Z_T, N, K);
        
        dim3 dimGrid4(N / BLOCK_SIZE + 1, 1 / BLOCK_SIZE + 1);
        matrix_mul_shared<<<dimGrid4,dimBlock>>>(d_yhat, d_Z, d_v, N, K, 1);
        
        // backwards:
        vector_sub<<<N / LINEAR_BLOCK_SIZE + 1, LINEAR_BLOCK_SIZE>>>(d_error, d_yhat, d_y, N);
        
        dim3 dimGrid5(K / BLOCK_SIZE + 1, 1 / BLOCK_SIZE + 1);
        matrix_mul_shared<<<dimGrid5,dimBlock>>>(d_grad_v, d_Z_T, d_error, K, N, 1);
        
        dim3 dimGrid6(N / BLOCK_SIZE + 1, K / BLOCK_SIZE + 1);
        d_relu_matrix_mul_shared<<<dimGrid6,dimBlock>>>(d_grad_Z, d_error, d_v, d_Z, N, 1, K);
        
        dim3 dimGrid7(N / BLOCK_SIZE + 1, K / BLOCK_SIZE + 1);
        matrix_transpose<<<dimGrid7,dimBlock>>>(d_grad_p_T, d_grad_Z, K, N);
        
        dim3 dimGrid8(K / BLOCK_SIZE + 1, D / BLOCK_SIZE + 1);
        matrix_mul_shared<<<dimGrid8,dimBlock>>>(d_grad_W, d_grad_p_T, d_X, K, N, D);
        
        // update
        update<<<N / LINEAR_BLOCK_SIZE + 1, LINEAR_BLOCK_SIZE>>>(d_W, d_grad_W, (STEP/N), W_N);
        update<<<N / LINEAR_BLOCK_SIZE + 1, LINEAR_BLOCK_SIZE>>>(d_v, d_grad_v, (STEP/N), V_N);
        // cudaMemcpy(h_W, d_W, sizeof(double) * W_N, cudaMemcpyDeviceToHost);
        // cudaMemcpy(h_v, d_v, sizeof(double) * V_N, cudaMemcpyDeviceToHost);

        // get MSE back
        square<<<N / LINEAR_BLOCK_SIZE + 1, LINEAR_BLOCK_SIZE>>>(d_err_sq, d_error, N);
        hipMemcpy(h_err_sq, d_err_sq, sizeof(double) * N, hipMemcpyDeviceToHost);
        double sum = 0.0;
        for(int i = 0; i < N; i++){
            sum += h_err_sq[i];
        }
        printf("MSE is %f\n", sum / N);
    }

    // Verification
    /*
    for(int i = 0; i < K; i++){
        for(int j = 0; j < D; j++){
            // double sum = 0.0;
            // for(int k = 0; k < A_WIDTH; k++){
            //     sum += h_A[i*A_WIDTH+k] * h_B[k*B_WIDTH + j];
            // }
            // h_ref[i * C_WIDTH + j] = sum;
            // assert(fabs(h_ref[i*C_WIDTH + j] - h_C[i * C_WIDTH + j]) < MAX_ERR);
            printf("h_W[%d][%d] = %f\n", i, j, h_W[i * D + j]);
            // printf("h_Z[%d][%d] = %f\n", i, j, h_Z[i * K + j]);
            // printf("h_ref[%d][%d] = %f\n", i, j, h_ref[i * C_WIDTH + j]);
        }
    }
    for(int i = 0; i < K; i++){
        printf("h_v[%d] = %f\n", i, h_v[i]);
    }
    */
    printf("PASSED\n");

    // Deallocate device memory
    hipFree(d_X);
    hipFree(d_X_T);
    hipFree(d_W);
    hipFree(d_v);
    hipFree(d_Z);
    hipFree(d_Z_T);
    hipFree(d_yhat);
    hipFree(d_y);
    hipFree(d_error);
    hipFree(d_grad_v);
    hipFree(d_grad_Z);
    hipFree(d_grad_p_T);
    hipFree(d_grad_W);
    hipFree(d_err_sq);

    // Deallocate host memory
    free(h_X); 
    free(h_W);
    free(h_v);
    free(h_Z);
    free(h_Z_T);
    free(h_yhat);
    free(h_y);
    free(h_error);
    free(h_grad_v);
    free(h_grad_Z);
    free(h_grad_p_T);
    free(h_grad_W);
    free(h_err_sq);
}
