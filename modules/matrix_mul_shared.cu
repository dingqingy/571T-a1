#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// matrix multiplication: C = AB
// A
#define A_HEIGHT 1024 
#define A_WIDTH 1024 
#define A_N A_HEIGHT * A_WIDTH 
// B
#define B_HEIGHT A_WIDTH
#define B_WIDTH 1024 
#define B_N B_HEIGHT * B_WIDTH

// C
#define C_HEIGHT A_HEIGHT
#define C_WIDTH B_WIDTH
#define C_N C_HEIGHT * C_WIDTH

#define BLOCK_SIZE 32
#define MAX_ERR 1e-6

__global__ void matrix_mul_shared(double *d_C, double *d_A, double *d_B, int d_a_height, int d_a_width, int d_b_width) {
    // global position in the C (output) matrix
    int cid = blockIdx.y * blockDim.y + threadIdx.y;
    int rid = blockIdx.x * blockDim.x + threadIdx.x;

    // block position
    int c_block = blockIdx.y;
    int r_block = blockIdx.x;

    // thread position in the block
    int c_thread = threadIdx.y;
    int r_thread = threadIdx.x;

    __shared__ double d_A_sub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double d_B_sub[BLOCK_SIZE][BLOCK_SIZE];

    double sum = 0.0;

    // iterate over tiles across horizontal direction of A
    for(k = 0; k<(d_a_width+1)/BLOCK_SIZE + 1; k++){
        // load d_A[rid, k*BLOCK_SIZE+c_thread] into d_A_sub[r_thread][c_thread]
        if((rid < d_a_height) && (k*BLOCK_SIZE+c_thread < d_a_width)){
            d_A_sub[r_thread][c_thread] = d_A[rid*d_a_width + k*BLOCK_SIZE+c_thread];
        } else {
            d_A_sub[r_thread][c_thread] = 0.0;
        }

        // load d_B[k*BLOCK_SIZE + r_thread, cid] into d_B_sub[r_thread][c_thread]
        if((k*BLOCK_SIZE+r_thread < d_a_width) && (cid < d_b_width)){
            d_B_sub[r_thread][c_thread] = d_B[(k*BLOCK_SIZE+r_thread)*d_b_width + cid];
        } else {
            d_B_sub[r_thread][c_thread] = 0.0;
        }

        __syncthreads();

        // dot product within a tile
        for(i = 0; i<d_a_width; i++){
            sum += d_A_sub[r_thread][i] * d_B_sub[i][c_thread];
        }

        __syncthreads();
    }

    // write sum back
    if(rid < d_a_height && cid < d_b_width)
        d_C[rid * d_b_width + cid] = sum;
}

// float float_rand( float min, float max )
// {
//     float scale = rand() / (float) RAND_MAX; 
//     return min + scale * ( max - min );      
// }

int main(){
    double *h_A, *h_B, *h_C;
    double *d_A, *d_B, *d_C;
    double *h_ref; // compute verified matMul
    // Allocate host memory
    h_A = (double*)malloc(sizeof(double) * A_N);
    h_B = (double*)malloc(sizeof(double) * B_N);
    h_C = (double*)malloc(sizeof(double) * C_N);
    h_ref = (double*)malloc(sizeof(double) * C_N);

    // Initialize host arrays
    
    /***       TEST 1    ***/
    // for(int i = 0; i < A_HEIGHT; i++){
    //     for(int j = 0; j < A_WIDTH; j++){
    //         h_A[i*A_WIDTH + j] = (float)i;
    //     }
    // }
    // for(int i = 0; i < B_HEIGHT; i++){
    //     for(int j = 0; j < B_WIDTH; j++){
    //         h_B[i*B_WIDTH + j] = (float)i;
    //     }
    // }
    
    srand((unsigned int)time(NULL));
    /***       TEST 2    ***/
    for (int i = 0; i< A_N; i++){
        h_A[i] = (double)rand()/(double)(RAND_MAX);
    }
    for (int i = 0; i< B_N; i++){
        h_B[i] = (double)rand()/(double)(RAND_MAX);
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, sizeof(double) * A_N);
    hipMalloc((void**)&d_B, sizeof(double) * B_N);
    hipMalloc((void**)&d_C, sizeof(double) * C_N);

    // Transfer data from host to device memory
    hipMemcpy(d_A, h_A, sizeof(double) * A_N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(double) * B_N, hipMemcpyHostToDevice);

    // Executing kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
    // Note C_mat row maps to x dimension, and col maps to y dimension
    dim3 dimGrid(C_HEIGHT / BLOCK_SIZE + 1, C_WIDTH / BLOCK_SIZE + 1);
    // dim3 dimGrid(2, 1);
    matrix_mul<<<dimGrid,dimBlock>>>(d_C, d_A, d_B, A_HEIGHT, A_WIDTH, B_WIDTH);
    
    // Transfer data back to host memory
    hipMemcpy(h_C, d_C, sizeof(double) * C_N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < C_HEIGHT; i++){
        for(int j = 0; j < C_WIDTH; j++){
            double sum = 0.0;
            for(int k = 0; k < A_WIDTH; k++){
                sum += h_A[i*A_WIDTH+k] * h_B[k*B_WIDTH + j];
            }
            h_ref[i * C_WIDTH + j] = sum;
            assert(fabs(h_ref[i*C_WIDTH + j] - h_C[i * C_WIDTH + j]) < MAX_ERR);
            // printf("h_c[%d][%d] = %f\n", i, j, h_C[i * C_WIDTH + j]);
            // printf("h_ref[%d][%d] = %f\n", i, j, h_ref[i * C_WIDTH + j]);
        }
    }
    printf("PASSED\n");

    // Deallocate device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Deallocate host memory
    free(h_A); 
    free(h_B);
    free(h_C);
}
