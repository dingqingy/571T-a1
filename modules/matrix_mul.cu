#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>


// matrix multiplication: C = AB
// A
#define A_HEIGHT 1024 
#define A_WIDTH 1024 
#define A_N A_HEIGHT * A_WIDTH 
// B
#define B_HEIGHT A_WIDTH
#define B_WIDTH 1024 
#define B_N B_HEIGHT * B_WIDTH

// C
#define C_HEIGHT A_HEIGHT
#define C_WIDTH B_WIDTH
#define C_N C_HEIGHT * C_WIDTH

#define BLOCK_SIZE 32
#define MAX_ERR 1e-6

__global__ void matrix_mul(double *d_C, double *d_A, double *d_B, int d_a_height, int d_a_width, int d_b_width) {
    int cid = blockIdx.y * blockDim.y + threadIdx.y;
    int rid = blockIdx.x * blockDim.x + threadIdx.x;

    if(rid < d_a_height && cid < d_b_width){
    // sum: to evaluated dot product
        double sum = 0.0;
        for(int k = 0; k < d_a_width; k++){
            sum += d_A[rid * d_a_width + k] * d_B[d_b_width*k + cid];
        }
        d_C[rid * d_b_width + cid] = sum;
    }
}

// float float_rand( float min, float max )
// {
//     float scale = rand() / (float) RAND_MAX; 
//     return min + scale * ( max - min );      
// }

int main(){
    double *h_A, *h_B, *h_C;
    double *d_A, *d_B, *d_C;
    double *h_ref; // compute verified matMul
    // Allocate host memory
    h_A = (double*)malloc(sizeof(double) * A_N);
    h_B = (double*)malloc(sizeof(double) * B_N);
    h_C = (double*)malloc(sizeof(double) * C_N);
    h_ref = (double*)malloc(sizeof(double) * C_N);

    // Initialize host arrays
    
    /***       TEST 1    ***/
    // for(int i = 0; i < A_HEIGHT; i++){
    //     for(int j = 0; j < A_WIDTH; j++){
    //         h_A[i*A_WIDTH + j] = (float)i;
    //     }
    // }
    // for(int i = 0; i < B_HEIGHT; i++){
    //     for(int j = 0; j < B_WIDTH; j++){
    //         h_B[i*B_WIDTH + j] = (float)i;
    //     }
    // }
    
    srand((unsigned int)time(NULL));
    /***       TEST 2    ***/
    for (int i = 0; i< A_N; i++){
        h_A[i] = (double)rand()/(double)(RAND_MAX);
    }
    for (int i = 0; i< B_N; i++){
        h_B[i] = (double)rand()/(double)(RAND_MAX);
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, sizeof(double) * A_N);
    hipMalloc((void**)&d_B, sizeof(double) * B_N);
    hipMalloc((void**)&d_C, sizeof(double) * C_N);

    // Transfer data from host to device memory
    hipMemcpy(d_A, h_A, sizeof(double) * A_N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(double) * B_N, hipMemcpyHostToDevice);

    // Executing kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
    // Note C_mat row maps to x dimension, and col maps to y dimension
    dim3 dimGrid(C_HEIGHT / BLOCK_SIZE + 1, C_WIDTH / BLOCK_SIZE + 1);
    // dim3 dimGrid(2, 1);
    matrix_mul<<<dimGrid,dimBlock>>>(d_C, d_A, d_B, A_HEIGHT, A_WIDTH, B_WIDTH);
    
    // Transfer data back to host memory
    hipMemcpy(h_C, d_C, sizeof(double) * C_N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < C_HEIGHT; i++){
        for(int j = 0; j < C_WIDTH; j++){
            double sum = 0.0;
            for(int k = 0; k < A_WIDTH; k++){
                sum += h_A[i*A_WIDTH+k] * h_B[k*B_WIDTH + j];
            }
            h_ref[i * C_WIDTH + j] = sum;
            assert(fabs(h_ref[i*C_WIDTH + j] - h_C[i * C_WIDTH + j]) < MAX_ERR);
            // printf("h_c[%d][%d] = %f\n", i, j, h_C[i * C_WIDTH + j]);
            // printf("h_ref[%d][%d] = %f\n", i, j, h_ref[i * C_WIDTH + j]);
        }
    }
    printf("PASSED\n");

    // Deallocate device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Deallocate host memory
    free(h_A); 
    free(h_B);
    free(h_C);
}
