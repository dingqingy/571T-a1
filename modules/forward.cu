#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>


// forward propogation
/*
def forward(X, W, v):
    Z_trans = relu(W@X.T) # mat-mat
    Z = Z_trans.T # trans
    yhat = Z@v # mat-vec
    return Z, yhat
*/

/* Parameter Setup */
#define N 4 // # of input samples
#define D 2 // # of input neurons
#define K 3 // # of hidden neurons

// X: input matrix (n * d)
#define X_HEIGHT N
#define X_WIDTH D
#define X_N X_HEIGHT * X_WIDTH
// Z: ifmap matrix (n * k)
#define Z_HEIGHT N
#define Z_WIDTH K
#define Z_N Z_HEIGHT * Z_WIDTH 
// W: layer 1 weights (k * d)
#define W_HEIGHT K
#define W_WIDTH D
#define W_N W_HEIGHT * W_WIDTH
// v: layer 2 weights
#define V_HEIGHT K
#define V_WIDTH 1
#define V_N V_HEIGHT * V_WIDTH

#define BLOCK_SIZE 32
#define MAX_ERR 1e-6

__global__ void matrix_mul(double *d_C, double *d_A, double *d_B, int d_a_height, int d_a_width, int d_b_width) {
    int cid = blockIdx.y * blockDim.y + threadIdx.y;
    int rid = blockIdx.x * blockDim.x + threadIdx.x;

    if(rid < d_a_height && cid < d_b_width){
    // sum: to evaluated dot product
        double sum = 0.0;
        for(int k = 0; k < d_a_width; k++){
            sum += d_A[rid * d_a_width + k] * d_B[d_b_width*k + cid];
        }
        d_C[rid * d_b_width + cid] = sum;
    }
}

__global__ void relu_matrix_mul(double *d_C, double *d_A, double *d_B, int d_a_height, int d_a_width, int d_b_width) {
    int cid = blockIdx.y * blockDim.y + threadIdx.y;
    int rid = blockIdx.x * blockDim.x + threadIdx.x;

    if(rid < d_a_height && cid < d_b_width){
    // sum: to evaluated dot product
        double sum = 0.0;
        for(int k = 0; k < d_a_width; k++){
            sum += d_A[rid * d_a_width + k] * d_B[d_b_width*k + cid];
        }
        d_C[rid * d_b_width + cid] = (sum>0)?sum:0;
    }
}

__global__ void matrix_transpose(double *d_out, double *d_in, int d_in_width, int d_out_width) {
    int cid = blockIdx.y * blockDim.y + threadIdx.y;
    int rid = blockIdx.x * blockDim.x + threadIdx.x;

    if(cid < d_in_width && rid < d_out_width){
        d_out[cid * d_out_width + rid] = d_in[rid * d_in_width + cid];
    }
}

int main(){
    double *h_X, *h_W, *h_v;
    double *h_Z, *h_Z_T, *h_yhat;
    double *d_X, *d_X_T, *d_W, *d_v;
    double *d_Z, *d_Z_T, *d_yhat;
    // double *h_ref; // compute verified results
    // Allocate host memory
    h_X = (double*)malloc(sizeof(double) * X_N);
    h_W = (double*)malloc(sizeof(double) * W_N);
    h_v = (double*)malloc(sizeof(double) * V_N);
    h_Z_T = (double*)malloc(sizeof(double) * Z_N);
    h_Z = (double*)malloc(sizeof(double) * Z_N);
    h_yhat = (double*)malloc(sizeof(double) * N);
    // h_ref = (double*)malloc(sizeof(double) * N);

    // Initialize host arrays
    
    /***       TEST 1    ***/
    for(int i = 0; i < X_N; i++){
        if(i == 1 || i == 3){
            h_X[i] = (double)(-i-1);
        } else{
            h_X[i] = (double)(i+1);
        }
    }
    for(int i = 0; i < W_N; i++){
        h_W[i] = double(i+1);
    }
    for(int i = 0; i < V_HEIGHT; i++){
        h_v[i] = (double)(i+1);
    }
    
    /***       TEST 2    ***/
    // rand((unsigned int)time(NULL));
    // for (int i = 0; i< A_N; i++){
    //     h_A[i] = (double)rand()/(double)(RAND_MAX);
    // }
    // for (int i = 0; i< B_N; i++){
    //     h_B[i] = (double)rand()/(double)(RAND_MAX);
    // }

    // Allocate device memory
    hipMalloc((void**)&d_X, sizeof(double) * X_N);
    hipMalloc((void**)&d_X_T, sizeof(double) * X_N);
    hipMalloc((void**)&d_Z, sizeof(double) * Z_N);
    hipMalloc((void**)&d_Z_T, sizeof(double) * Z_N);
    hipMalloc((void**)&d_W, sizeof(double) * W_N);
    hipMalloc((void**)&d_v, sizeof(double) * V_N);
    hipMalloc((void**)&d_yhat, sizeof(double) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_X, h_X, sizeof(double) * X_N, hipMemcpyHostToDevice);
    hipMemcpy(d_W, h_W, sizeof(double) * W_N, hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, sizeof(double) * V_N, hipMemcpyHostToDevice);

    // Executing kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    // X_HEIGHT (N) corresponding to OUT_WIDTH, X_WIDTH (D) corresponding to IN_WIDTH
    dim3 dimGrid1(N / BLOCK_SIZE + 1,D / BLOCK_SIZE + 1);
    matrix_transpose<<<dimGrid1,dimBlock>>>(d_X_T, d_X, D, N);
    
    dim3 dimGrid2(K / BLOCK_SIZE + 1, N / BLOCK_SIZE + 1);
    relu_matrix_mul<<<dimGrid2,dimBlock>>>(d_Z_T, d_W, d_X_T, K, D, N);
    
    dim3 dimGrid3(K / BLOCK_SIZE + 1, N / BLOCK_SIZE + 1);
    matrix_transpose<<<dimGrid3,dimBlock>>>(d_Z, d_Z_T, N, K);
    
    dim3 dimGrid4(N / BLOCK_SIZE + 1, 1 / BLOCK_SIZE + 1);
    matrix_mul<<<dimGrid4,dimBlock>>>(d_yhat, d_Z, d_v, N, K, 1);
    // Transfer data back to host memory
    hipMemcpy(h_yhat, d_yhat, sizeof(double) * N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < N; i++){
        for(int j = 0; j < 1; j++){
            // double sum = 0.0;
            // for(int k = 0; k < A_WIDTH; k++){
            //     sum += h_A[i*A_WIDTH+k] * h_B[k*B_WIDTH + j];
            // }
            // h_ref[i * C_WIDTH + j] = sum;
            // assert(fabs(h_ref[i*C_WIDTH + j] - h_C[i * C_WIDTH + j]) < MAX_ERR);
            printf("h_yhat[%d][%d] = %f\n", i, j, h_yhat[i * 1 + j]);
            // printf("h_ref[%d][%d] = %f\n", i, j, h_ref[i * C_WIDTH + j]);
        }
    }
    printf("PASSED\n");

    // Deallocate device memory
    hipFree(d_X);
    hipFree(d_X_T);
    hipFree(d_W);
    hipFree(d_v);
    hipFree(d_Z);
    hipFree(d_Z_T);
    hipFree(d_yhat);

    // Deallocate host memory
    free(h_X); 
    free(h_W);
    free(h_v);
    free(h_Z);
    free(h_Z_T);
    free(h_yhat);
}
